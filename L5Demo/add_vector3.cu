
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float* a, float* b, float* c)
{
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   c[idx] = a[idx] + b[idx];
}

int main()
{
   int N = 1024;
   int numBytes = N * sizeof(float);
   float* h_a = (float *)malloc(numBytes);
   float* h_b = (float *)malloc(numBytes);
   float* h_c = (float *)malloc(numBytes);

   for (int i=0; i<N; ++i)
   {
      h_a[i] = i;
      h_b[i] = i;
      h_c[i] = 0;
   }

   float *d_a, *d_b, *d_c;
   hipError_t errCode = hipMalloc((float**)&d_a, numBytes); 
   if (errCode != hipSuccess)
   {
      printf("hipMalloc Failed\n");
      exit(EXIT_FAILURE);
   }

   errCode = hipMalloc((float**)&d_b, numBytes);
   if (errCode != hipSuccess) 
   {
           printf("hipMalloc returned %d\n-> %s\n",
                        static_cast<int>(errCode), hipGetErrorString(errCode));
           exit(EXIT_FAILURE);
   }
   

   hipMalloc((float**)&d_c, numBytes);

   hipMemcpy(d_a, h_a, numBytes, hipMemcpyHostToDevice);
   hipMemcpy(d_b, h_b, numBytes, hipMemcpyHostToDevice);

   add<<<1, N>>>(d_a, d_b, d_c);
   
   hipDeviceSynchronize();
 
   errCode = hipGetLastError();
   if (errCode != hipSuccess) 
   {
           printf("kernet launch returned %d\n-> %s\n",
                        static_cast<int>(errCode), hipGetErrorString(errCode));
           exit(EXIT_FAILURE);
   }

   hipMemcpy(h_c, d_c, numBytes, hipMemcpyDeviceToHost);

   for (int i=0; i<N; ++i)
   {
     printf("%d: %f\n", i, h_c[i]);
   }

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   free(h_a);
   free(h_b);
   free(h_c);

   return 0;
}
