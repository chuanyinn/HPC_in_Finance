
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float* a, float* b, float* c, int N)
{
   int stride = blockDim.x * gridDim.x;
   int threadId = threadIdx.x + blockIdx.x * blockDim.x;
   for (int idx=threadId; idx<N; idx+= stride)
       c[idx] = a[idx] + b[idx];
}

int main()
{
   int N = 1024;
   int numBytes = N * sizeof(float);
   float* h_a = (float *)malloc(numBytes);
   float* h_b = (float *)malloc(numBytes);
   float* h_c = (float *)malloc(numBytes);

   for (int i=0; i<N; ++i)
   {
      h_a[i] = i;
      h_b[i] = i;
      h_c[i] = 0;
   }

   float *d_a, *d_b, *d_c;
   hipMalloc((float**)&d_a, numBytes); 
   hipMalloc((float**)&d_b, numBytes);
   hipMalloc((float**)&d_c, numBytes);

   hipMemcpy(d_a, h_a, numBytes, hipMemcpyHostToDevice);
   hipMemcpy(d_b, h_b, numBytes, hipMemcpyHostToDevice);

   add<<<2, 256>>>(d_a, d_b, d_c, N);
   hipDeviceSynchronize();

   hipMemcpy(h_c, d_c, numBytes, hipMemcpyDeviceToHost);

   for (int i=0; i<N; ++i)
   {
     printf("%d: %f\n", i, h_c[i]);
   }

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   free(h_a);
   free(h_b);
   free(h_c);

   return 0;
}
