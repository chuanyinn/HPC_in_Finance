/* * This program uses the host CURAND API to generate 100 
   pseudorandom floats. */ 

//taken from the following and modified slightly
//http://docs.nvidia.com/cuda/curand/host-api-overview.html#host-api-example 

#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand.h> 

int main(int argc, char *argv[]) 
{ 
   size_t N = 100; 
   hiprandGenerator_t gen; 
   float *devData, *hostData; 

   int numBytes = N*sizeof(float);

   /* Allocate n floats on host */ 
   hostData = (float *)malloc(numBytes); 

   /* Allocate n floats on device */ 
   hipMalloc((float**)&devData, numBytes); 

   /* Create pseudo-random number generator */ 
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  
   /* Set seed */ 
   hiprandSetPseudoRandomGeneratorSeed(gen, 1ULL); 
  
   /* Generate n floats on device */ 
   hiprandGenerateUniform(gen, devData, N); 
  
   /* Copy device memory to host */ 
   hipMemcpy(hostData, devData, numBytes, 
         hipMemcpyDeviceToHost); 
   
   /* Show result */ 
   for(int i = 0; i < 10; i++) 
   { 
        printf("%1.4f\n", hostData[i]); 
   } 

   /* Cleanup */ 
   hiprandDestroyGenerator(gen); 
  
   hipFree(devData); 
   free(hostData); 

} 

